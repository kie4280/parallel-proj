#include <hip/hip_runtime.h>
#include<stdio.h>
#define BOARD_LENGTH 64

__global__ void leaf_simple_evaluation(char *chess_raw, float *scores, int n) {
    int id = threadIdx.x;
    float result = 0.0f;
    if (id < n){
        int lower = id * BOARD_LENGTH;

        for(int i = 0; i < BOARD_LENGTH; i++){
            char temp = chess_raw[lower + i];
            if(temp == ' ')
                continue;
            else if(temp == 'K')
                result += 200;
            else if(temp == 'k')
                result -= 200;
            else if(temp == 'Q')
                result += 9;
            else if(temp == 'q')
                result -= 9;
            else if(temp == 'R')
                result += 5;
            else if(temp == 'r')
                result -= 5;
            else if(temp == 'B')
                result += 3;
            else if(temp == 'b')
                result -= 3;
            else if(temp == 'N')
                result += 3;
            else if(temp == 'n')
                result -= 3;
            else if(temp == 'P')
                result += 1;
            else if(temp == 'p')
                result -= 1;
        }
        scores[id] = 1.0;
    }
};
//BUG: in leaf_simple_evaluation nothing happened

void hostFE(char* chess_raw, float* scores, int n)
{
    float *scores_d;
    char *chess_raw_d;
    hipMalloc((void **)&chess_raw_d, n * BOARD_LENGTH * sizeof(char));
    hipMalloc((void **)&scores_d, n * sizeof(float));


    hipMemcpy(chess_raw_d, chess_raw, n * BOARD_LENGTH * sizeof(char), hipMemcpyHostToDevice);


    dim3 thread_num(128);
    dim3 block_num((n + thread_num.x - 1) / thread_num.x);
    leaf_simple_evaluation<<<block_num ,thread_num>>>(chess_raw_d, scores_d, n);
    hipDeviceSynchronize();


    hipMemcpy(scores, scores_d, n * sizeof(float), hipMemcpyDeviceToHost);


    hipFree(chess_raw_d);
    hipFree(scores_d);
};
