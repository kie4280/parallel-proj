#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "leaf.h"

__global__ void leaf_simple_evaluation(char *chess_raw, float *scores, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int result = 0;

    if (id < n){
        char *p = chess_raw + id * BOARD_LENGTH;

        for(int i = 0; i < BOARD_LENGTH; i++){
            if(p[i] == '0')
                continue;
            else if(p[i] == 'K')
                result += 200;
            else if(p[i] == 'k')
                result -= 200;
            else if(p[i] == 'Q')
                result += 9;
            else if(p[i] == 'q')
                result -= 9;
            else if(p[i] == 'R')
                result += 5;
            else if(p[i] == 'r')
                result -= 5;
            else if(p[i] == 'B')
                result += 3;
            else if(p[i] == 'b')
                result -= 3;
            else if(p[i] == 'N')
                result += 3;
            else if(p[i] == 'n')
                result -= 3;
            else if(p[i] == 'P')
                result += 1;
            else if(p[i] == 'p')
                result -= 1;
        }
        scores[id] = result / 20.0;
    }
    
}

void hostFE(char *chess_raw, float *scores, int n)
{
    float *scores_d;
    char *chess_raw_d;
    hipMalloc((void **)&chess_raw_d, n * BOARD_LENGTH * sizeof(char));
    hipMalloc((void **)&scores_d, n * sizeof(float));

    //move data from h to d
    hipMemcpy(chess_raw_d, chess_raw, n * BOARD_LENGTH * sizeof(char), hipMemcpyHostToDevice);

    //kernel function 1-D
    int blockSize = 1024;
    int gridSize = (int)ceil((float)n/blockSize);
    leaf_simple_evaluation<<<blockSize, gridSize>>>(chess_raw_d, scores_d, n);

    //move data from d to h
    hipDeviceSynchronize();
    hipMemcpy(scores, scores_d, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(chess_raw_d);
    hipFree(scores_d);
}
