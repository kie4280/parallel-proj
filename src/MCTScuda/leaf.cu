#include <hip/hip_runtime.h>
#include <stdio.h>
#define BOARD_LENGTH 64

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void leaf_simple_evaluation(char *chess_raw, float *scores, int n) {
    int id = threadIdx.x;
    float result = 0.0f;
    if (id < n){
        int lower = id * BOARD_LENGTH;

        for(int i = 0; i < BOARD_LENGTH; i++){
            char temp = chess_raw[lower + i];
            if(temp == ' ')
                continue;
            else if(temp == 'K')
                result += 200;
            else if(temp == 'k')
                result -= 200;
            else if(temp == 'Q')
                result += 9;
            else if(temp == 'q')
                result -= 9;
            else if(temp == 'R')
                result += 5;
            else if(temp == 'r')
                result -= 5;
            else if(temp == 'B')
                result += 3;
            else if(temp == 'b')
                result -= 3;
            else if(temp == 'N')
                result += 3;
            else if(temp == 'n')
                result -= 3;
            else if(temp == 'P')
                result += 1;
            else if(temp == 'p')
                result -= 1;
        }
        scores[id] = result;
    }
}
//BUG: in leaf_simple_evaluation nothing happened, scores is always 0.

void hostFE(char* chess_raw, float* scores, int n)
{
    float *scores_d;
    char *chess_raw_d;
    hipMalloc((void **)&chess_raw_d, n * BOARD_LENGTH * sizeof(char));
    hipMalloc((void **)&scores_d, n * sizeof(float));

    hipMemcpy(chess_raw_d, chess_raw, n * BOARD_LENGTH * sizeof(char), hipMemcpyHostToDevice);

    int thread_num = 128;
    int block_num = (n + thread_num - 1) / thread_num;
    leaf_simple_evaluation<<<block_num, thread_num>>>(chess_raw_d, scores_d, n);
    hipDeviceSynchronize();

    hipMemcpy(scores, scores_d, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(chess_raw_d);
    hipFree(scores_d);
}
